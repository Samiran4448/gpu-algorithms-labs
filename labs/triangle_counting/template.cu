#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

__device__ uint64_t Intersect_Lin(const uint32_t *const edgeDst, size_t srcStart, size_t srcEnd, size_t dstStart, size_t dstEnd) {
  uint64_t TC = 0;
  if (srcStart < srcEnd && dstStart < dstEnd) {
    uint32_t W1 = edgeDst[srcStart];
    uint32_t W2 = edgeDst[dstStart];
    while (srcStart < srcEnd && dstStart < dstEnd) {
      if (W1 < W2) {
        W1 = edgeDst[++srcStart];
      } else if (W1 > W2) {
        W2 = edgeDst[++dstStart];
      } else if (W1 == W2) {
        W1 = edgeDst[++srcStart];
        W2 = edgeDst[++dstStart];
        ++TC;
      }
    }
    return TC;
  } else {
    return 0;
  }
}

__global__ static void kernel_tc(uint64_t *__restrict__ triangleCounts, //!< per-edge triangle counts
                                 const uint32_t *const edgeSrc,         //!< node ids for edge srcs
                                 const uint32_t *const edgeDst,         //!< node ids for edge dsts
                                 const uint32_t *const rowPtr,          //!< source node offsets in edgeDst
                                 const size_t numEdges                  //!< how many edges to count triangles for
) {
  size_t edgeID = threadIdx.x + blockIdx.x * blockDim.x;
  if (edgeID < numEdges && numEdges > 0) {
    // Determine the source and destination node for the edge
    uint32_t src = edgeSrc[edgeID];
    uint32_t dst = edgeDst[edgeID];

    // Use the row pointer array to determine the start and end of the neighbor list in the column index array
    size_t srcStart = rowPtr[src];
    size_t srcEnd   = (numEdges > rowPtr[src + 1]) ? rowPtr[src + 1] : numEdges;

    size_t dstStart = rowPtr[dst];
    size_t dstEnd   = (numEdges > rowPtr[dst + 1]) ? rowPtr[dst + 1] : numEdges;
    // Determine how many elements of those two arrays are common

    triangleCounts[edgeID] = Intersect_Lin(edgeDst, srcStart, srcEnd, dstStart, dstEnd);
  }
}

uint64_t count_triangles(const pangolin::COOView<uint32_t> view, const int mode) {
  //@@ create a pangolin::Vector (uint64_t) to hold per-edge triangle counts
  // Pangolin is backed by CUDA so you do not need to explicitly copy data between host and device.
  // You may find pangolin::Vector::data() function useful to get a pointer for your kernel to use.

  uint64_t total = 0;

  //@@ calculate the number of blocks needed
  // dim3 dimGrid (ceil(number of non-zeros / dimBlock.x))

  //@@ create a pangolin::Vector (uint64_t) to hold per-edge triangle counts
  // Pangolin is backed by CUDA so you do not need to explicitly copy data between host and device.
  // You may find pangolin::Vector::data() function useful to get a pointer for your kernel to use.
  pangolin::Vector<uint64_t> edge_counts = pangolin::Vector<uint64_t>(view.nnz(), 0);
  //@@ launch the linear search kernel here
  dim3 dimBlock(1024);
  if (mode == 1) {
    dim3 dimGrid((view.nnz() + dimBlock.x - 1) / dimBlock.x);
    // std::cout << view.nnz() << " view.nnz() test" << std::endl;
    // std::cout << dimGrid.x << " Grid dimensions test" << std::endl;
    kernel_tc<<<dimGrid, dimBlock>>>(edge_counts.data(), view.row_ind(), view.col_ind(), view.row_ptr(), view.nnz());
    hipDeviceSynchronize();  
    // for (int i = 0; i < 10; i++) {
    //   printf("count %d\t", (int) edge_counts.data()[i]);
    //   printf("Edge source %d\t", view.row_ind()[i]);
    //   printf("Edge destination %d\n", view.col_ind()[i]);
    // }
    // printf("testing row_ptr array\n");
    // for (int i = 0; i < 10; i++) {
    //   printf("row ptr for %d\t to %d\n", view.row_ptr()[i], view.row_ptr()[i + 1]);
    // }

    // printf("row ptr from %d\t to %d\n", view.row_ptr()[1], view.row_ptr()[2]);
    // printf("Code reached here %d\n", __LINE__);

    //@@ do a global reduction (on CPU or GPU) to produce the final triangle count

  } else if (mode == 2) {

    //@@ launch the hybrid search kernel here
    // your_kernel_name_goes_here<<<dimGrid, dimBlock>>>(...)

  } else {
    assert("Unexpected mode");
    return uint64_t(-1);
  }

  
  //@@ do a global reduction (on CPU or GPU) to produce the final triangle count
  // CPU reduction
  for (uint64_t i = 0; i < view.nnz(); ++i) {
    total += edge_counts.data()[i];
  }
  // printf("code reached here %d\n", __LINE__);

  // return total
  return total;

}