#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

__device__ uint64_t Intersect_Lin(const uint32_t *const edgeDst, size_t srcStart, size_t srcEnd, size_t dstStart, size_t dstEnd) {
  uint64_t TC = 0;
  if (srcStart < srcEnd && dstStart < dstEnd) {
    uint32_t W1 = edgeDst[srcStart];
    uint32_t W2 = edgeDst[dstStart];
    while (srcStart < srcEnd && dstStart < dstEnd) {
      if (W1 < W2) {
        W1 = edgeDst[++srcStart];
      }
      if (W1 > W2) {
        W2 = edgeDst[++dstStart];
      }
      if (W1 == W2) {
        W1 = edgeDst[++srcStart];
        W2 = edgeDst[++dstStart];
        ++TC;
      }
    }
    return TC;
  } else
    return 0;
}

__global__ static void kernel_tc(uint64_t *__restrict__ triangleCounts, //!< per-edge triangle counts
                                 const uint32_t *const edgeSrc,         //!< node ids for edge srcs
                                 const uint32_t *const edgeDst,         //!< node ids for edge dsts
                                 const uint32_t *const rowPtr,          //!< source node offsets in edgeDst
                                 const size_t numEdges                  //!< how many edges to count triangles for
) {
  size_t edgeID = threadIdx.x + blockIdx.x * blockDim.x;
  if (edgeID < numEdges) {
    // Determine the source and destination node for the edge
    uint32_t src = edgeSrc[edgeID];
    uint32_t dst = edgeDst[edgeID];

    // Use the row pointer array to determine the start and end of the neighbor list in the column index array
    size_t srcStart = rowPtr[src];
    size_t srcEnd   = rowPtr[src+1];

    size_t dstStart = rowPtr[dst];
    size_t dstEnd   = rowPtr[dst + 1];
    // Determine how many elements of those two arrays are common

    triangleCounts[edgeID] = Intersect_Lin(edgeDst, srcStart, srcEnd, dstStart, dstEnd);
  }
}

uint64_t count_triangles(const pangolin::COOView<uint32_t> view, const int mode) {
  if (mode == 1) {

    // REQUIRED

    //@@ create a pangolin::Vector (uint64_t) to hold per-edge triangle counts
    // Pangolin is backed by CUDA so you do not need to explicitly copy data between host and device.
    // You may find pangolin::Vector::data() function useful to get a pointer for your kernel to use.
    pangolin::Vector<uint64_t> edge_counts;
    //@@ launch the linear search kernel here
    dim3 dimBlock(512);
    dim3 dimGrid((view.nnz() + dimBlock.x - 1) / dimBlock.x);
    std::cout << view.nnz() << " view.nnz() test" << std::endl;
    std::cout << dimGrid.x << " Grid dimensions test" << std::endl;
    kernel_tc<<<dimGrid, dimBlock>>>(edge_counts.data(), view.row_ind(), view.col_ind(), view.row_ptr(), view.nnz());

    printf("Code reached here\n");
    uint64_t total = 0;
    //@@ do a global reduction (on CPU or GPU) to produce the final triangle count
    //CPU reduction
    for (uint64_t i = 0; i < view.nnz(); i++) {
      total += edge_counts.data()[i];
    }
    printf("code reached here too\n");
    return total;

  } else if (2 == mode) {

    // OPTIONAL. See README for more details

    uint64_t total = 0;
    //@@ do a global reduction (on CPU or GPU) to produce the final triangle count

    return total;
  } else {
    assert("Unexpected mode");
    return uint64_t(-1);
  }
}
