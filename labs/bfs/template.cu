#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 4096

// Number of warp queues per block
#define NUM_WARP_QUEUES 8
// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY (BQ_CAPACITY / NUM_WARP_QUEUES)

/******************************************************************************
 GPU kernels
*******************************************************************************/

__global__ void gpu_global_queueing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to global queue
}

__global__ void gpu_block_queueing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE

  // Initialize shared memory queue (size should be BQ_CAPACITY)

  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to block queue
  // If full, add neighbor to global queue

  // Allocate space for block queue to go into global queue

  // Store block queue in global queue
}

__global__ void gpu_warp_queueing_kernel(unsigned int *nodePtrs,
                                        unsigned int *nodeNeighbors,
                                        unsigned int *nodeVisited,
                                        unsigned int *currLevelNodes,
                                        unsigned int *nextLevelNodes,
                                        unsigned int *numCurrLevelNodes,
                                        unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  // This version uses NUM_WARP_QUEUES warp queues of capacity 
  // WQ_CAPACITY.  Be sure to interleave them as discussed in lecture.  

  // Don't forget that you also need a block queue of capacity BQ_CAPACITY.

  // Initialize shared memory queues (warp and block)

  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to the queue
  // If full, add neighbor to block queue
  // If full, add neighbor to global queue

  // Allocate space for warp queue to go into block queue

  // Store warp queues in block queue (use one warp or one thread per queue)
  // Add any nodes that don't fit (remember, space was allocated above)
  //    to the global queue

  // Saturate block queue counter (too large if warp queues overflowed)
  // Allocate space for block queue to go into global queue

  // Store block queue in global queue
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                        unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes,
                        unsigned int *numCurrLevelNodes,
                        unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queueing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                       unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes,
                       unsigned int *numCurrLevelNodes,
                       unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queueing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                      unsigned int *nodeVisited, unsigned int *currLevelNodes,
                      unsigned int *nextLevelNodes,
                      unsigned int *numCurrLevelNodes,
                      unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queueing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}
