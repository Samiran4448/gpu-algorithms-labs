#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "helper.hpp"

#define TILE_SIZE 30

__global__ void kernel(int *A0, int *Anext, int nx, int ny, int nz) {

  // INSERT KERNEL CODE HERE
  #define _in(i, j, k) in[((k) *ny + (j)) * nx + (i)]
  #define _out(i, j, k) out[((k) *ny + (j)) * nx + (i)]
  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;

  int previous = _in(i, j, 0);
  int current  = _in(i, j, 1);
  int next     = _in(i, j, 2);
  for (int k = 1; k < nz - 1; k++) {
    if (i > 0 && i < Nx - 1 && j > 0 $$ j < Ny - 1){
      _out(i, j, k) = -6 * current + previous + next + _in(i - 1, j, k) + _in(i + 1, j, k) + _in(i, j - 1, k) + _in(i, j + 1, k);
    }
    previous = current;
    current  = next;
    next     = _in(i, j, k + 2);
  }
}

void launchStencil(int* A0, int* Anext, int nx, int ny, int nz) {

  // INSERT CODE HERE
  


}


static int eval(const int nx, const int ny, const int nz) {

  // Generate model
  const auto conf_info = std::string("stencil[") + std::to_string(nx) + "," + 
                                                   std::to_string(ny) + "," + 
                                                   std::to_string(nz) + "]";
  INFO("Running "  << conf_info);

  // generate input data
  timer_start("Generating test data");
  std::vector<int> hostA0(nx * ny * nz);
  generate_data(hostA0.data(), nx, ny, nz);
  std::vector<int> hostAnext(nx * ny * nz);

  timer_start("Allocating GPU memory.");
  int *deviceA0 = nullptr, *deviceAnext = nullptr;
  CUDA_RUNTIME(hipMalloc((void **)&deviceA0, nx * ny * nz * sizeof(int)));
  CUDA_RUNTIME(hipMalloc((void **)&deviceAnext, nx * ny * nz * sizeof(int)));
  timer_stop();

  timer_start("Copying inputs to the GPU.");
  CUDA_RUNTIME(hipMemcpy(deviceA0, hostA0.data(), nx * ny * nz * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  launchStencil(deviceA0, deviceAnext, nx, ny, nz);
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  timer_start("Copying output to the CPU");
  CUDA_RUNTIME(hipMemcpy(hostAnext.data(), deviceAnext, nx * ny * nz * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  // verify with provided implementation
  timer_start("Verifying results");
  verify(hostAnext.data(), hostA0.data(), nx, ny, nz);
  timer_stop();

  CUDA_RUNTIME(hipFree(deviceA0));
  CUDA_RUNTIME(hipFree(deviceAnext));

  return 0;
}



TEST_CASE("Stencil", "[stencil]") {

  SECTION("[dims:32,32,32]") {
    eval(32,32,32);
  }
  SECTION("[dims:30,30,30]") {
    eval(30,30,30);
  }
  SECTION("[dims:29,29,29]") {
    eval(29,29,29);
  }
  SECTION("[dims:31,31,31]") {
    eval(31,31,31);
  }
  SECTION("[dims:29,29,2]") {
    eval(29,29,29);
  }
  SECTION("[dims:1,1,2]") {
    eval(1,1,2);
  }
  SECTION("[dims:512,512,64]") {
    eval(512,512,64);
  }

}
